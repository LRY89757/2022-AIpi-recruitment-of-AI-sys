#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>


__global__ void Convolution(float* kernel, float* im, float*dst, 
                        int width, int height, int inChannel,
                        int kernelH, int kernelW, int outChannel, 
                        int stride, int padd)
{
    // to do
}

int main()
{
    int width = 256, height = 256, channels = 3, kernelsize = 3, 
                    stride = 1, outChannel = 256, int padd = 0,
                    kernelH = kernelsize, kernelW = kernelsize;

    float *im = new float[height * width * channels];

    for (int ch = 0; ch < channels; ch++)
    {
        for (int row = 0; row < height; row++)
        {
            for (int col = 0; col < width; col++)
            {
                im[ch * height * width + row * width + col] = (row * width + col);
            }
        }
    }

    float *kernel = new float[outChannel * kernelH * kernelW];
    for(int ch = 0;ch<outChannel;ch++)
    {
        int dst = ch + 1;
        float* src0 = kernel + ch*kernelH*kernelW;
        for(int i = 0;i<kernelH*kernelW;i++)
        {
            src0[i] = dst;
        }
    }

    // To do
    // timing
    // Convolution func
    // timing

    return 0;
}